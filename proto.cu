#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include <time.h> 
#include <string.h>

__global__ double diff_centrada(double f, float coef_a,int idx, float h){
  double df;
  for (int m=0;m<11;m++){
    df += coef_a[m]*f[idx-5+m];
  }
  return df/h;
}
__global__ double diff_atrasada(double f, float coef_a,int idx, float h){
  double df;
  for (int m=-10;m<1;m++){
    df += coef_a[m+10]*f[idx+m];
  }
  return df/h;
}
__global__ double diff_adelantada(double f, float coef_a,int idx, float h,){
  double df;
  for (int m=-idx;m<11;m++){
    df += coef_a[m+idx]*f[idx+m];
  }
  return df/h;
}
__global__ void diff()
__global__ double Runge_Kutta4_temporal( double func,double func_past, double yn, double h, int Nr){
  int id =threadIdx.x + blockDim.x*blockIdx.x;

  double k1 = h * func_past;
  double k2_3 = h*(func_past + 0.5* (func_past-func));  //interpolacion lineal
  double k4 = h*(func);  
  double func_plus;
  yn_plus = y_n + 1.0/6.0 * (k1+4.0*k2_3+k4);
  return yn_plus
}
__global__ void evolution_phi( double phi, double t, double pi, double alpha, double A, double B, double dx, double dt, int Nr){
  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    phi[ (t+1) * Nr + idx] =( alpha[idx] /( sqrtd(A[idx]) * B[idx] )) * dt + phi[ t * Nr + idx];
}
__global__ void evolution_phi( double chi, double t, double pi, double alpha, double A, double B, double dx, double dt, int Nr){
  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func=( alpha[idx] /( sqrtd(A[idx]) * B[idx] )) * dt + phi[ t * Nr + idx];
    dr_func=diff_centrada(func,coef_a,idx,dr);
    chi[ (t+1) * Nr + idx] =Runge_Kutta4();
}
__global__ void Kb_dot( double Kb, double K, double A,double B, double alpha, double Db, double Da,
  double lamba, double U, double rho, double S_A, double dx, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func,func_1,func_2,func_3, dr_Db;

    func_1 = 0.5 * U + 2.0 * lambda * B / A - Db - lambda - Da;
    dr_Db = 0;
    func_2 = -0.5 * Da * Db - 0.5*dr_Db + 0.25*Db*(U + 4.0*lambda*B/A) +A*K*Kb;
    func_3 = S_A - rho;

    func= alpha/(A* (idx*dr) )*func_1 - alpha/A*func_2 + alpha/(2.0)*func_3;


    Kb=Runge_Kutta4(func);
}

__global__ void K_dot( double K, double Kb, double A,double B, double alpha, double Db, double Da,
  double lamba, double U, double rho, double S_A, double dx, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func,func_1,func_2,func_3, dr_Da;

    func_1 = K*K - 4.0*K*Kb + 6*Kb*Kb;
    dr_Da = 0;
    func_2 = Da*Da + dr_Da + 2.0*Da/( idx*dr ) + 0.5*Da*(U + 4.0*lambda*B/A);
    func_3 = rho + S_A + 2.0*S_B;

    func= alpha*func_1 - alpha/A*func_2 + alpha/(2.0)*func_3;


    Kb=Runge_Kutta4(func);
}

__global__ void lambda_dot( double lamba, double Kb, double K,double A, double B, double alpha, double Db,
  double ja, float dr, float dt, int Nr, int t){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
  if(t==0){
   lambda[idx]=(1.0-A[idx]/B[idx])/(idx*dr)
  }
  else{
    double func, dr_Kb;
    dr_Kb = 0;

    func= 2.0*alpha*A/B*(dr_Kb - 0.5*Db*( K - 3.0*Kb ) + 0.5*ja);


    Kb=Runge_Kutta4(func);
  }
}

__global__ void U_dot( double U, double Kb, double K, double A, double B, double alpha, double Db,
  double Da, double lambda, double ja, float dx, float dt, int Nr, int t){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
  if(t==0){
   U[idx]=(1.0-4.0*lambda[idx]/A[idx]);
  }
  else{
    double func,func_1,func_2,func_3, dr_Kb;
    dr_K = 0;
    func_1 = dr_K + Da*(K - 4.0*Kb);
    func_2 = 2.0*(K -3.0*Kb)*(Db - 2.0*lambda*B/A);
    func_3 = 4.0*alpha*ja;

    func= -2.0*alpha * (func_1 - func_2) - func_3;


    Kb=Runge_Kutta4(func);
  }
}

__global__ void A_dot( double A, double K, double Kb, double alpha, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func, dt_A,sum_A;
    dt_A = 0;
    sum_A = 0;
    func= -2.0*alpha*A*(K - 2.0*Kb);
    float b_i[4]={1.0,2.0,2.0,1.0};
    flaot c_i[4]={1.0,0.5,0.5,1.0};
    float a_ij[4][4]={{0,0,0,0},{0.5,0,0,0},{0,0.5,0,0},{0,0,1.0,0}};
    float k_i[4];
    double temp,temp2,k_i,temp_sum;
    temp2=0;
    temp_sum=temp;
    temp = -2.0*alpha[idx]*(K[idx] - 2.0*Kb[idx]);
    
    for (int i=0;i<4;i++){
     temp2=A[idx];
     for(int j=0;j<i;j++){
      temp2 += a_ij[i][j]*k_i[j];}
     k_i[i]=temp*temp2;
     temp_sum += b[i]*k_i[i]*h*1.0/6.0;}
    A[idx] = A[idx] + temp_sum;//pasó temporal
}

__global__ void B_dot( double B,double Kb, double alpha, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func;
    dr_K = 0;


    func= -2.0*alpha*B*Kb;
    B=Runge_Kutta4(func);
}

__global__ void Db_dot( double B,double Kb, double alpha, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double dr_func;
    func=alpha*Kb;
    dr_func= 0;
    B=Runge_Kutta4(-2.0*dr_func);
}

__global__ void alpha_dot( double B,double Kb, double alpha, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func,f_alpha;
    f_alpha=0;
    func=alpha*alpha*f_alpha*K;
    
    B=Runge_Kutta4(-func);
}

__global__ void Da_dot( double B,double Kb, double alpha, double dt, int Nr){

  int idx =threadIdx.x + blockDim.x*blockIdx.x;
    double func,f_alpha, dr_func;
    f_alpha=0;
    func=alpha*f_alpha*K;
    dr_func=0
    B=Runge_Kutta4(-dr_func);
}
__global__ void calculate_rho(double rho,double PI,double chi, double A, double B, int Nr, int t ){
   int idx;
   rho=(PI[idx]*PI[idx] / (B[idx]*B[idx]) + chi[idx]*chi[idx])/(2.0*A[idx]);
}
__global__ void calculate_ja(double ja,double PI,double chi, double A, double B, int Nr, int t ){
   int idx;
   ja = -PI[idx]*chi[idx] / (sqrt(A[idx])*B[idx]);
}
__global__ void calculate_SA(double SA,double PI,double chi, double A, double B, int Nr, int t ){
   int idx;
   SA=(PI[idx]*PI[idx] / (B[idx]*B[idx]) + chi[idx]*chi[idx])/(2.0*A[idx]);
}
__global__ void calculate_SB(double SB,double PI,double chi, double A, double B, int Nr, int t ){
   int idx;
   SB=(PI[idx]*PI[idx] / (B[idx]*B[idx]) - chi[idx]*chi[idx])/(2.0*A[idx]);
}
  //Resolver el albertiano(phi) = 0
  //phi_tplus=
  //Nota: Deja todas las variables sin [], despues lo remplazas
__global__ void phi_evolution(double phi, double phi_t_plus, double pi, double A, double B, double alpha, float dt, int Nr, int t){
idx= threadlock.x +
If (idx <Nr){
phi[ t * Nr + idx ] = pi[idx] * alpha[idx]/(sqrtf(A[idx])*B[idx]) * dt + phi[ t * Nr + idx ];}
}

__global__ void chi_evolution( double chi, double A , double B, double alpha, float dt , int Nr){
idx=
If (idx<Nr){
If (idx=Nr-1){
}
else if (idx=0){
}
phi_dot_plus=alpha[ idx+1 ]*pi[ idx+1 ]/(sqrtd(A[ idx+1 ])*B[ idx+1 ]);
phi_dot_minus=alpha[ idx-1 ]*pi[ idx-1 ]/(sqrtd(A[ idx-1 ])*B[ idx-1 ]);
chi[idx]=( phi_dot_plus - phi_dot_minus )/(2.0*dr) *dt +chi[idx]; }
}
 void incial_phi(double phi, float dr,int Nr){
  float a=0.02;
  float std=0.15;
  for (int i=0;i<Nr;i++){
   phi[i]=a*exp(-(i*dr/std)*(i*dr/std))
  }
 }
void incial_chi(double chi, double phi, float dr,int Nr){
 for(int i=0;i<Nr;i++){
  chi[i]=difference_tenth(phi,dr,i);
 }
}
void rellenar(double f, int Nr,double num){
 for (int i=0;i<Nr;i++){
  f[i]=num;
 }
}
int main(){
 int Nr=1000;
 int Nt=10000;
// Defino los array del host
 double A,B,alpha,phi,chi,PI,lambda,K,Kb,U;
//Defino los array de device
 double cuda_A,cuda_B,cuda_alpha,cuda_phi,cuda_chi,cuda_PI,cuda_K,cuda_Kb,cuda_U;

//deltas
 float dr=1.0/Nr;
 float dt=1.0/Nt;

// mallocs
  A=(double)malloc(Nr*sizeof(double));


//condiciones iniciales
inicial_phi(phi,dr,Nr);
inicial_chi(chi,phi,dr,Nr);
rellenar(PI, Nr, 0);
rellenar(K,Nr,0.0);
rellenar(Kb,Nr,0.0);
rellenar(Da,Nr,0.0);
rellenar(Db,Nr,0.0);
rellenar(alpha, Nr, 1.0);
rellenar(B,Nr,1.0);
//pendiente inicial del A...
// cuda mallocs

  hipMalloc ((void**)cuda_A, Nr*sizeof(double) );

//mmcopy
 hipMemcpy( cuda_phi, phi, Nr*sizeof(double), hipMemcpyHostToDevice );
 for(int t=0; t<Nt;t++){


 }

 hipMemcpy( phi, cuda_phi, Nr*sizeof(double), hipMemcpyDeviceToHost );




  free(phi);free(chi);free(PI);free(K)free(Kb);free(U),free(A);free(B);free(alpha);free(lambda);
  hipFree(cuda_phi);hipFree(cuda_chi);hipFree(cuda_A);hipFree(cuda_B);hipFree(cuda_alpha);
  hipFree(cuda_K);cuadFree(cuda_Kb);hipFree(cuda_lambda);hipFree(cuda_U);

}
